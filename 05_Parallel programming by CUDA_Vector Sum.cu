#include <iostream>

#include "hip/hip_runtime.h"



static void HandleError(hipError_t, const char*, int);
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define N 10

__global__ void add(int* a, int* b, int* c)
{
	int tid = blockIdx.x;	// Compute the data of this index.
	if (tid < N)
		c[tid] = a[tid] + b[tid];
}


int main(void)
{
	int a[N], b[N], c[N];
	int* dev_a, * dev_b, * dev_c;

	// allocate GPU memory
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));

	// fill 'a' and 'b' in CPU
	for (int i = 0; i < N; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}

	// copy 'a' and 'b' to GPU memory
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

	add <<<N, 1 >>> (dev_a, dev_b, dev_c);

	// Copy the result of 'add', 'c' from GPU memory to Host(CPU) memory
	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

	// output result
	for (int i = 0; i < N; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	// deallocate GPU memory
	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_c));

	return 0;
}


// ���� �߻��� ��� �� �����ϴ� �Լ� - å ������ ����.
static void HandleError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}