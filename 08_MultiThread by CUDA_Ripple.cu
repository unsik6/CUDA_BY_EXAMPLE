#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "...\CUDA-training-master\utils\cuda_by_example\common\cpu_bitmap.h"
#include "...\CUDA-training-master\utils\cuda_by_example\common\cpu_anim.h"


static void HandleError(hipError_t, const char*, int);
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define DIM 1024

struct DataBlock
{
	unsigned char* dev_bitmap;
	CPUAnimBitmap* bitmap;
};

// GPU에 할당한 메모리 해제
void cleanup(DataBlock* d)
{
	hipFree(d->dev_bitmap);
}

__global__ void kernel(unsigned char* ptr, int ticks)
{
	// threadIdx/blockIdx로 픽셀 위치를 결정한다.
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	// 이제 해당 위치의 값을 계산한다.
	float fx = x - DIM / 2;
	float fy = y - DIM / 2;
	float d = sqrtf(fx * fx + fy * fy);
	unsigned char grey = (unsigned char)(128.0f + 127.0f * cos(d / 10.0f - ticks / 7.0f) / (d / 10.0f + 1.0f));

	ptr[offset * 4 + 0] = grey;
	ptr[offset * 4 + 1] = grey;
	ptr[offset * 4 + 2] = grey;
	ptr[offset * 4 + 3] = 255;
}

void generate_frame(DataBlock* d, int ticks)
{
	dim3 blocks(DIM / 16, DIM / 16);
	dim3 threads(16, 16);

	kernel <<<blocks, threads >>> (d->dev_bitmap, ticks);

	HANDLE_ERROR(hipMemcpy(d->bitmap->get_ptr(), d->dev_bitmap, d->bitmap->image_size(), hipMemcpyDeviceToHost));
}

int main(void)
{
	DataBlock data;
	
	// 1024 * 1024 크기의 애니메이션 비트맵을 생성한다.
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;

	// 디바이스 메모리에 image 크기만큼 할당
	HANDLE_ERROR(hipMalloc((void**)&data.dev_bitmap, bitmap.image_size()));

	bitmap.anim_and_exit((void(*)(void*, int)) generate_frame, (void(*)(void*)) cleanup);

}

// 에러 발생시 출력 후 종료하는 함수 - 책 예제에 포함.
static void HandleError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}
