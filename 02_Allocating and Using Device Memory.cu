#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

// ���� �߻��� ��� �� �����ϴ� �Լ� - å ������ ����.
static void HandleError(hipError_t, const char*, int);
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void add(int a, int b, int* c) {
	*c = a + b;
}

int main(void)
{
	int c;
	int* dev_c;

	// __host___device__ hipError_t hipMalloc(void** ���� �Ҵ��� �޸��� �ּҸ� ����Ű�� ������, size_t �Ҵ��� �޸��� ũ��)
	/* -CUDA ��Ÿ���� ����̽� �޸𸮸� �Ҵ��Ѵ�.�ش� �޸𸮸� ����Ű�� �����ʹ� ù��° ���ڿ� ����ȴ�.
	* - return: hipError_t ����ü
	* 	 0: hipSuccess: ����.
	* 	 1: hipErrorInvalidValue: �Ķ������ ���� ������ �ʰ��Ͽ� ����.
	* 	 2: hipErrorOutOfMemory: �Ҵ��� �޸� ������ �ʹ� �۾Ƽ� ����.
	*  
	*  - ȣ��Ʈ���� ����Ǵ� �ڵ忡�� hipMalloc()�� ���� ��ȯ�Ǵ� ������(����̽� ������)�� �������ؼ��� �ȵȴ�.
	* 	 �������� ��ġ �̵�, �����͸� �̿��� ����, �������� Ÿ�� ��ȯ�� ����
	* 	 �������� �޸𸮸� �аų� ����ϱ� ���� �� �� ����.
	*  - hipMalloc()���� �Ҵ��� �޸� �����͸� ����̽����� ����Ǵ� �Լ��� ������ �� �ִ�.
	* 	 ����̽����� ����Ǵ� �ڵ忡�� hipMalloc()���� �Ҵ��� �޸� �����͸� �̿��Ͽ� �޸𸮸� �аų� �� �� �ִ�.
	* 	 hipMalloc()���� �Ҵ��� �޸� �����͸� ȣ��Ʈ���� ����Ǵ� �Լ��� ������ �� �ִ�. �ٸ�, ȣ��Ʈ���� ����Ǵ� �ڵ忡�� hipMalloc()���� �Ҵ��� �޸� �����͸� �̿��Ͽ� �аų� �� �� ����.
	*/
	HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));


	// ����̽� �޸𸮿� �����ϴ� �������� ��� 1
	add <<<1, 1 >> > (2, 7, dev_c);

	// ����̽� �޸𸮿� �����ϴ� �������� ��� 2
	// __host__ hipError_t hipMemcpy(src, dst, hipMemcpyKind)
	/* ����° ���ڴ� src�� dst�� ���� ��� ���ϴ��� ��Ÿ����.
	* - return: hipMalloc�� ����
	* - hipMemcpyKind ����ü
	*	0: hipMemcpyHostToHost: ȣ��Ʈ���� ȣ��Ʈ	// ��, �� ��쿡�� �׳� C�� memcpy�� ����ϸ� �ȴ�.
	*	1: hipMemcpyHostToDevice: ȣ��Ʈ���� ����̽�
	*	2: hipMemcpyDeviceToHost: ����̽����� ȣ��Ʈ
	*	3: cudaMecmcpyDeviceToDevice: ����̽����� ����̽�
	*	4: hipMemcpyDefault: src �����Ͱ� �����̳Ŀ� ���� �߷еȴ�.
	* 
	*/
	HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));

	printf("2 + 7 = %d\n", c);

	// hipMalloc()���� �Ҵ��� �޸𸮴� C�� free()�� ������ �� ����
	// hipFree()�� ���� �����Ѵ�.
	hipFree(dev_c);

	return 0;
}

static void HandleError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}