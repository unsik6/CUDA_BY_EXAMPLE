#include "hip/hip_runtime.h"
#include <iostream>

#ifndef __HIPCC__
#define __HIPCC__
	#include <hip/device_functions.h>
#endif

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "E:\00_NEW_ERA\01_INHA\00_TCLAB\07_CUDA\CUDA_PRACTICE_01\CUDA_PRACTICE_01\CUDA-training-master\utils\cuda_by_example\common\cpu_bitmap.h"
#include "E:\00_NEW_ERA\01_INHA\00_TCLAB\07_CUDA\CUDA_PRACTICE_01\CUDA_PRACTICE_01\CUDA-training-master\utils\cuda_by_example\common\cpu_anim.h"


static void HandleError(hipError_t, const char*, int);
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define imin(a, b) (a < b ? a : b)

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(float* a, float* b, float* c)
{
	// ���� �޸��� ���, �ϳ��� Block �ȿ� �����ϴ� ��� Thread���� �����ϴ� �޸��̴�.
	// �Ʒ��� �ڵ忡���� �����޸��� cache �迭(Block �� Thread ���� ��ŭ�� ����)�� �� ��ҵ��� �� Thread�� ����Ѵ�.
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;

	// �� Thread���� ���� ������ ���� ������ �յ��� ��� ���Ѵ�.
	float temp = 0;
	while (tid < N)
	{
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	// �� Thread���� �����޸� �迭�� ��� �� �ڽ��� ����ϴ� ��ҿ� ���� ���� �κ� ���� �����Ѵ�.
	cache[cacheIndex] = temp;

	// �� Block�� Thread���� ����ȭ��Ų��.
	// #ifndef __HIPCC__
	// #define __HIPCC__
	//		#include <hip/device_functions.h>
	// #endif
	// �� ��ó�� ������ �־���� �Ѵ�.
	__syncthreads();

	// ������
	// ���� �ڵ� ������, �������� ���ؼ��� threadPerBlock�� 2�� ������� �Ѵ�.
	int i = blockDim.x / 2;
	while (i != 0)
	{
		if (cacheIndex < i)
			cache[cacheIndex] += cache[cacheIndex + i];

		// �������� �� ������ ������ ������ �Ǿ����� ����ȭ�� �ʿ��ϴ�.
		// * ���� ����ȭ�� ���ؼ� �� ����ȭ �Լ��� ���� if �� �ȿ� �ִ´ٸ�, �ϳ��� ������ Ȱ���Ͽ� SW������ ����ȭ�� �����ϴ� ������� ���ؼ� Progress ���谡 ��Ÿ���� �Ͱ� ���� ������ �߻��ϰ� �ȴ�.
		__syncthreads();

		i /= 2;
	}

	if (cacheIndex == 0)
		c[blockIdx.x] = cache[0];
}

int main(void)
{
	float* a, * b, c, * partial_c;
	float* dev_a, * dev_b, * dev_partial_c;

	// CPU���� ����� �޸𸮸� �Ҵ��Ѵ�.
	a = (float*)malloc(N * sizeof(float));
	b = (float*)malloc(N * sizeof(float));
	partial_c = (float*)malloc(blocksPerGrid * sizeof(float));	// �κ� ���� ��� ��� ������ ���Ǿ� ����Ǹ� ������ Grid�� ����ϱ� ������ Grid �� Block�� �� ��ŭ �ʿ��ϴ�.

	// GPU ����̽� �޸𸮸� �Ҵ��Ѵ�.
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_partial_c, blocksPerGrid * sizeof(float))); // �κ� ���� ��� ��� ������ ���Ǿ� ����Ǹ� ������ Grid�� ����ϱ� ������ Grid �� Block�� �� ��ŭ �ʿ��ϴ�.

	// ȣ��Ʈ �޸𸮿� �����͸� ä���.
	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = 2 * i;
	}

	// ����̽� �޸𸮿� ȣ��Ʈ �޸𸮸� �����Ѵ�.
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice));

	// ���� ����
	dot << <blocksPerGrid, threadsPerBlock >> > (dev_a, dev_b, dev_partial_c);

	// �κ��� �迭�� ����̽����� ȣ��Ʈ�� �����Ѵ�.
	HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float),hipMemcpyDeviceToHost));

	// CPU���� �κ��� �迭�� ���� ���� ������ ����� ����Ѵ�.
	// �� ������ �������� ���� ��������, CPU���� �����ϴ� ������ GPU�� ����ϱ⿡�� ������ ���� �ſ� ���� �����̴�.
	c = 0;
	for (int i = 0; i < blocksPerGrid; i++)
	{
		c += partial_c[i];
	}

#define sum_squares(x) (x * (x + 1) * (2 * x + 1) / 6)
	printf("Does GPU value %.6g = %6.g?\n", c, 2 * sum_squares((float)(N - 1)));

	// �޸� ����
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_partial_c);
	free(a);
	free(b);
	free(partial_c);

	return 0;
}

// ���� �߻��� ��� �� �����ϴ� �Լ� - å ������ ����.
static void HandleError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}